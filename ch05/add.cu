#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <iostream>

const int N = 10;

__global__ void add(int *a, int *b, int *c){

    int tid = threadIdx.x;

    if(tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void){

    int a[N], b[N], c[N];

    int *dev_a, *dev_b, *dev_c;

    hipMalloc( (void**)&dev_a, sizeof(int) * N );
    hipMalloc( (void**)&dev_b, sizeof(int) * N);
    hipMalloc( (void**)&dev_c, sizeof(int) * N );

    for(int i = 0; i < N; i++){
        a[i] = -i;
        b[i] = i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++)
        std::cout << a[i] << std::endl;
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}