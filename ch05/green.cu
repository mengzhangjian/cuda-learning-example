#include "hip/hip_runtime.h"
//
// Created by zhangjian on 19-6-13.
//
#include "../common/book.h"
#include "../common/image.h"


const int DIM = 1024;
const float PI(3.1415926535897932f);
__global__ void kernel(unsigned char *ptr){


    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int offset = x + y * blockDim.x * gridDim.x;

    __shared__ float shared[16][16];

    const float period = 128.0f;
    shared[threadIdx.x][threadIdx.y] =
            255 * (sinf(x * 2.0f * PI / period) + 1.0f) *
                    (sinf(y * 2.0f * PI / period) + 1.0f) / 4.0f;
    __syncthreads();

    ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = shared[15 - threadIdx.x][15 - threadIdx.y];
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}


int main(void){

    IMAGE bitmap(DIM, DIM);
    unsigned char *dev_bitmap;
    dim3 grid(DIM / 16, DIM / 16);
    dim3 threads(16, 16);

    hipMalloc( (void**)&dev_bitmap, bitmap.image_size());

    kernel<<<grid, threads>>>(dev_bitmap);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap,
             bitmap.image_size(),
             hipMemcpyDeviceToHost);
    bitmap.show_image();
    hipFree(dev_bitmap);

}
