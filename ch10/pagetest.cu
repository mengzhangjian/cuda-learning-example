//
// Created by zhangjian on 19-6-22.
//
#include "../common/book.h"
#include "hip/hip_runtime.h"
#include <iostream>
#define  SIZE (10*1024*1024)
float cuda_malloc_test(int size, bool up){


    int *a, *dev_a;
    float elapsedTime;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (int*)malloc(size * sizeof(*a));
    hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

    hipEventRecord(start, 0);
    for(int i = 0; i < 100; i++){

        if(up)
            hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
        else
            hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    free(a);
    hipFree(dev_a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

float cuda_malloc_host_test(int size, bool up){

    int *a, *dev_a;
    float elapsedTime;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostAlloc( (void**)&a, size * sizeof(*a), hipHostMallocDefault);
    hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

    hipEventRecord(start, 0);
    for(int i = 0; i < 100; i++){

        if(up)
            hipMemcpy(dev_a, a, size * sizeof(*a), hipMemcpyHostToDevice);
        else
            hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipHostFree(a);
    hipFree(dev_a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

int main(void){

    float elapsedTime;
    float MB = (float)100*SIZE* sizeof(int)/1024/1024;
    elapsedTime = cuda_malloc_test(SIZE, true);

    std::cout<<elapsedTime<<std::endl;
    elapsedTime = cuda_malloc_host_test(SIZE, true);
    std::cout<<elapsedTime<<std::endl;
}