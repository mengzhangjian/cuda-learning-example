#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <iostream>
#include "hip/hip_runtime.h"
#include <math.h>
#include "../common/image.h"

using namespace cv;
using namespace std;


__global__ void color_to_gray(unsigned char *dev_bitmap, unsigned char *host_out, int width, int height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if(col < width && row < height){

        int grayoffset = row * width + col;

        int rgboffset = grayoffset * 3;

        unsigned char r = dev_bitmap[rgboffset + 1];
        unsigned char g = dev_bitmap[rgboffset + 2];
        unsigned char b = dev_bitmap[rgboffset + 3];
        host_out[grayoffset] = 0.21f * r + 0.71f * g + 0.07f * b;

    }

}

int main(void){

    Mat image = imread("av.jpg");
    Mat out = Mat::zeros(image.rows, image.cols, CV_8UC1);
    unsigned char *dev_bitmap;
    unsigned char *dev_out;
    hipMalloc((void**)&dev_bitmap, image.rows * image.cols * 3);
    hipMemcpy(dev_bitmap, image.data, image.rows * image.cols * 3, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_out, image.rows * image.cols);

    dim3 BlocksPerGrid(ceil(image.cols / 16.0), ceil(image.rows / 16.0), 1);
    dim3 ThreadsPerBlock(16, 16, 1);
    color_to_gray<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_bitmap, dev_out, image.cols, image.rows);
    hipMemcpy(out.data, dev_out, image.rows * image.cols, hipMemcpyDeviceToHost);
    imwrite("blur.jpg", out);
    hipFree(dev_bitmap);
    hipFree(dev_out);
    return 0;

}

